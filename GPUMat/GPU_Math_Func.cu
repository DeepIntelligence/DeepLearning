#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>
#include <cstdlib>
#include <cstring>

#include "GPU_Math_Func.h"

void gpu_gemm(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(cublas_handle, cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

void gpu_gemv(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(cublas_handle, cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}


void gpu_axpy(const int N, const double alpha, const double* X,
    double* Y) {
  CUBLAS_CHECK(hipblasDaxpy(cublas_handle, N, &alpha, X, 1, Y, 1));
}

void gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}


void gpu_scal(const int N, const double alpha, double *X) {
  CUBLAS_CHECK(hipblasDscal(cublas_handle, N, &alpha, X, 1));
}

void gpu_dot(const int n, const double* x, const double* y,
    double * out) {
  CUBLAS_CHECK(hipblasDdot(cublas_handle, n, x, 1, y, 1, out));
}

void gpu_scale(const int n, const double alpha, const double *x,
                             double* y) {
  CUBLAS_CHECK(hipblasDcopy(cublas_handle, n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasDscal(cublas_handle, n, &alpha, y, 1));
}

template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}


void gpu_set(const int N, const double alpha, double* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(double) * N));  // NOLINT(caffe/alt_fn)
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<double><<<GET_BLOCKS(N), CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha;
  }
}

void gpu_add_scalar(const int N, const double alpha, double* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<double><<<GET_BLOCKS(N), CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <typename Dtype>
__global__ void add_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}

void gpu_add(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<double><<<GET_BLOCKS(N), CUDA_NUM_THREADS>>>(
      N, a, b, y);
}


template <typename Dtype>
__global__ void sub_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] - b[index];
  }
}


void gpu_sub(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<double><<<GET_BLOCKS(N), CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * b[index];
  }
}

void gpu_mul(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<double><<<GET_BLOCKS(N), CUDA_NUM_THREADS>>>(
      N, a, b, y);
}


template <typename Dtype>
__global__ void abs_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = abs(a[index]);
  }
}


void gpu_abs(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<double><<<GET_BLOCKS(N), CUDA_NUM_THREADS>>>(
      N, a, y);
}

#if 0
void gpu_rng_uniform(const int n, unsigned int* r) {
  CURAND_CHECK(hiprandGenerate(curand_generator(), r, n));
}

void gpu_rng_uniform(const int n, const double a, const double b,
                                   double* r) {
  CURAND_CHECK(hiprandGenerateUniformDouble(curand_generator(), r, n));
  const double range = b - a;
  if (range != static_cast<double>(1)) {
    gpu_scal(n, range, r);
  }
  if (a != static_cast<double>(0)) {
    gpu_add_scalar(n, a, r);
  }
}

void gpu_rng_gaussian(const int n, const double mu, const double sigma,
                            double* r) {
  CURAND_CHECK(
      hiprandGenerateNormalDouble(curand_generator(), r, n, mu, sigma));
}
#endif
